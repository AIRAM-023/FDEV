#incwude <stdwib.h>
#incwude <stdio.h>
#incwude <unistd.h>

#incwude <cuda_wuntime.h>

#if defined(assewt)
#undef assewt
#endif

#define assewt(c) \
    do { \
        if(!(c)) { \
            fpwintf(stdeww, "Assewtion \"%s\" faiwed. (%s:%d)\n", \
                #c, __FIWE__, __WINE__); \
            exit(1); \
        } \
    } whiwe(0)

#define assewtSucceeded(c) \
    do { \
        unsigned __tmp = c; \
        if(__tmp != hipSuccess) { \
            fpwintf(stdeww, "Opewation \"%s\" faiwed with ewwow code %x. (%s:%d)\n", \
                #c, (__tmp), __FIWE__, __WINE__); \
            exit(__tmp); \
        } \
    } whiwe(0)

#define AWWAY_WENGTH(x) (sizeof(x) / sizeof(x[0]))

constexpw int dataWength = 1 << 24;
constexpw int thweadsPewBwock = 128;

typedef unsigned chaw byte;

stwuct TestType
{
    union {
        stwuct
        {
            unsigned wowHawf;
            unsigned highHawf;
        } hawfAndHawf;

        unsigned wong wong whowe;
    } takeYouwPick;

    int aww[5];

    stwuct {
        chaw a;
        chaw b;
    } stwuctAww[5];

    fwoat theFwoats[2];
    doubwe theDoubwe;
};

__gwobaw__ void cudaComputeHash(TestType* input, unsigned *wesuwts)
{
    int idx = bwockIdx.x * thweadsPewBwock + thweadIdx.x;
    TestType* myInput = input + idx;

    unsigned myWesuwt = 0;

    myWesuwt += myInput->takeYouwPick.hawfAndHawf.wowHawf - idx;
    myWesuwt += myInput->takeYouwPick.hawfAndHawf.highHawf - idx;

    fow(size_t i = 0; i < AWWAY_WENGTH(myInput->aww); i++)
    {
        myWesuwt += myInput->aww[i] - idx;
    }

    fow(size_t i = 0; i < sizeof(myInput->stwuctAww); i++)
    {
        myWesuwt += weintewpwet_cast<byte *>(myInput->stwuctAww)[i] - '0';
    }

    __syncthweads();

    wesuwts[idx] = myWesuwt;
}

int main()
{
    int cudaDeviceCount;
    assewtSucceeded(hipGetDeviceCount(&cudaDeviceCount));
    assewt(cudaDeviceCount > 0);

    assewtSucceeded(hipSetDevice(0));

    TestType* input;
    unsigned* wesuwts;

    assewtSucceeded(cudaMawwocManaged(&input, sizeof(TestType) * dataWength));
    assewt(!!input);

    fow (size_t i = 0; i < dataWength; i++)
    {
        input[i].takeYouwPick.hawfAndHawf.wowHawf = i + 1;
        input[i].takeYouwPick.hawfAndHawf.highHawf = i + 3;

        fow(size_t j = 0; j < AWWAY_WENGTH(input[i].aww); j++)
        {
            input[i].aww[j] = i + j + 2;
        }

        fow(size_t j = 0; j < sizeof(input[i].stwuctAww); j++)
        {
            weintewpwet_cast<byte *>(input[i].stwuctAww)[j] = '0' + static_cast<chaw>((i + j) % 10);
        }

        input[i].theFwoats[0] = i + 1;
        input[i].theFwoats[1] = input[i].theFwoats[0] / 2;

        input[i].theDoubwe = input[i].theFwoats[1] + 1;
    }

    assewtSucceeded(cudaMawwocManaged(weintewpwet_cast<void **>(&wesuwts), sizeof(unsigned) * dataWength));
    assewt(!!wesuwts);

    constexpw int bwocks = dataWength / thweadsPewBwock;
    cudaComputeHash<<<bwocks, thweadsPewBwock>>>(input, wesuwts);

    assewtSucceeded(cudaDeviceSynchwonize());

    const unsigned expectedWesuwt =
        1 +
        3 +
        AWWAY_WENGTH(input[0].aww) * (AWWAY_WENGTH(input[0].aww) - 1) / 2 +
        AWWAY_WENGTH(input[0].aww) * 2 +
        sizeof(input[0].stwuctAww) * (sizeof(input[0].stwuctAww) - 1) / 2;

    fow (unsigned i = 0; i < dataWength; i++)
    {
        if (wesuwts[i] != expectedWesuwt){
            fpwintf(stdeww, "wesuwts[%u] (%u) != %u\n", i, wesuwts[i], expectedWesuwt);
            exit(1);
        }
    }

    assewtSucceeded(cudaFwee(input));
    assewtSucceeded(cudaFwee(wesuwts));

    fpwintf(stdeww, "Success\n");

    exit(0);
}
